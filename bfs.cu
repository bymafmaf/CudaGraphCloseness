#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <iostream>

#include "graphio.h"
#include "graph.h"

char gfile[2048];

using namespace::std;

void usage(){
  printf("./bfs <filename> <sourceIndex>\n");
  exit(0);
}
void random_ints(int* a, int N)
{
   int i;
   for (i = 0; i < N; ++i)
    a[i] = rand();
}
// CUDA STARTS

#define N (2048*2048)
#define M 512


__global__
void run(etype *row_ptr, vtype *col_ind, int nov);


/*
You can ignore the ewgths and vwghts. They are there as the read function expects those values
row_ptr and col_ind are the CRS entities. nov is the Number of Vertices
*/

int main(int argc, char *argv[]) {
  etype *row_ptr;
  vtype *col_ind;
  ewtype *ewghts;
  vwtype *vwghts;
  vtype nov;

  if(argc != 2)
  usage();

  const char* fname = argv[1];
  strcpy(gfile, fname);

  if(read_graph(gfile, &row_ptr, &col_ind, &ewghts, &vwghts, &nov, 0) == -1) {
    printf("error in graph read\n");
    exit(1);
  }
  /****** YOUR CODE GOES HERE *******/
  etype *d_row_ptr;
  vtype *d_col_ind;
  int size = nov * sizeof(int);

  hipMalloc((void **)&d_row_ptr, size);
  hipMalloc((void **)&d_col_ind, size);

  hipMemcpy(d_row_ptr, row_ptr, size, hipMemcpyHostToDevice);
  hipMemcpy(d_col_ind, col_ind, size, hipMemcpyHostToDevice);

  run<<<(N + M - 1)/M, M>>>(d_row_ptr, d_col_ind, nov);

  //hipMemcpy(c, dc, size, hipMemcpyDeviceToHost);

  // for (size_t i = 0; i < N; i++) {
  //   cout << c[i] << endl;
  // }
  hipFree(d_row_ptr);
  hipFree(d_col_ind);
  //hipFree(dc);

  free(row_ptr);
  free(col_ind);

  return 1;
}
