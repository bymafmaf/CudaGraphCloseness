#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <iostream>

#include "graphio.h"
#include "graph.h"

const int N = 512;

char gfile[2048];

using namespace::std;

void usage(){
  printf("./bfs <filename> <sourceIndex>\n");
  exit(0);
}
void random_ints(int* a, int N)
{
   int i;
   for (i = 0; i < N; ++i)
    a[i] = rand();
}
// CUDA STARTS



__global__
void run(etype *row_ptr, vtype *col_ind, etype *results, int nov);


/*
You can ignore the ewgths and vwghts. They are there as the read function expects those values
row_ptr and col_ind are the CRS entities. nov is the Number of Vertices
*/

int main(int argc, char *argv[]) {
  etype *row_ptr;
  vtype *col_ind;
  ewtype *ewghts;
  vwtype *vwghts;
  vtype nov;

  if(argc != 2)
  usage();

  const char* fname = argv[1];
  strcpy(gfile, fname);

  if(read_graph(gfile, &row_ptr, &col_ind, &ewghts, &vwghts, &nov, 0) == -1) {
    printf("error in graph read\n");
    exit(1);
  }
  /****** YOUR CODE GOES HERE *******/
  if (nov > N) {
    nov = N;
  }
  etype *d_row_ptr;
  vtype *d_col_ind;

  etype *d_results;
  int row_size = nov * sizeof(etype);
  int col_size = row_ptr[nov-1] * sizeof(vtype);

  hipMalloc((void **)&d_row_ptr, row_size);
  hipMalloc((void **)&d_col_ind, col_size);
  hipMalloc((void **)&d_results, row_size);

  hipMemcpy(d_row_ptr, row_ptr, row_size, hipMemcpyHostToDevice);
  hipMemcpy(d_col_ind, col_ind, col_size, hipMemcpyHostToDevice);

  printf("%s %d\n", "nov is", nov);
  run<<<(nov + 511)/512, 512>>>(d_row_ptr, d_col_ind, d_results, nov);

  etype *results = (etype *) malloc(row_size);
  hipMemcpy(results, d_results, row_size, hipMemcpyDeviceToHost);


  unsigned int max = 0;
  int maxIndex = -1;

  unsigned int totalNonZero = 0;
  for (unsigned int i = 0; i < nov; i++) {
    if (results[i] == 0) {
      continue;
    }
    totalNonZero++;
    if(results[i] >= max){
      max = results[i];
      maxIndex = i;
    }
  }
  cout << "total non zeroes " << totalNonZero << endl;
  printf("%s %d %s%d\n", "Min closeness belongs to", maxIndex, "with score of 1/", max);

  hipFree(d_row_ptr);
  hipFree(d_col_ind);
  //hipFree(dc);

  free(row_ptr);
  free(col_ind);

  return 1;
}
