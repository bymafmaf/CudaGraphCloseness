#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <iostream>

#include "graphio.h"
#include "graph.h"

char gfile[2048];

using namespace::std;

void usage(){
  printf("./bfs <filename> <sourceIndex>\n");
  exit(0);
}

// CUDA STARTS

const int N = 7;
const int blocksize = 7;

__global__
void add(int *da, int *db, int *dc);


/*
You can ignore the ewgths and vwghts. They are there as the read function expects those values
row_ptr and col_ind are the CRS entities. nov is the Number of Vertices
*/

int main(int argc, char *argv[]) {
  etype *row_ptr;
  vtype *col_ind;
  ewtype *ewghts;
  vwtype *vwghts;
  vtype nov;

  if(argc != 2)
  usage();

  const char* fname = argv[1];
  strcpy(gfile, fname);

  if(read_graph(gfile, &row_ptr, &col_ind, &ewghts, &vwghts, &nov, 0) == -1) {
    printf("error in graph read\n");
    exit(1);
  }
  /****** YOUR CODE GOES HERE *******/
  int a, b, c;
  int *da, *db, *dc;
  int size = sizeof(int);

  hipMalloc((void **)&da, size);
  hipMalloc((void **)&db, size);
  hipMalloc((void **)&dc, size);

  a = 5;
  b = 10;
  hipMemcpy(da, &a, size, hipMemcpyHostToDevice);
  hipMemcpy(db, &b, size, hipMemcpyHostToDevice);

  add<<<1,1>>>(da, db, dc);

  hipMemcpy(&c, dc, size, hipMemcpyDeviceToHost);

  printf("%d\n", c);
  hipFree(da);
  hipFree(db);
  hipFree(dc);

  free(row_ptr);
  free(col_ind);

  return 1;
}
