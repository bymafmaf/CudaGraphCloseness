#include "hip/hip_runtime.h"
#include <stdio.h>
#include "graph.h"
#include <queue>

//etype unsigned int

__device__
void recur(etype vertice, etype level, etype *visited, etype *row_ptr, vtype *col_ind, etype *results, int nov){
  if (vertice == nov-1) {
    return;
  }
  for (size_t i = row_ptr[vertice]; i < row_ptr[vertice+1]; i++) {
    etype n_ind = col_ind[i];
    if (visited[n_ind] == 0) {
      visited[n_ind] = level;
    }
  }
  for (size_t i = row_ptr[vertice]; i < row_ptr[vertice+1]; i++) {
    etype n_ind = col_ind[i];
    if (visited[n_ind] == level) {
      recur(n_ind, level+1, visited, row_ptr, col_ind, results, nov);
    }
  }
}

__global__
void run(etype *row_ptr, vtype *col_ind, etype *results, int nov) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < nov) {
    etype *visited = new etype[nov];
    for (size_t i = 0; i < nov; i++) {
      visited[i] = 0;
    }

    recur(index, 1, visited, row_ptr, col_ind, results, nov);

    etype distanceSum = 0;
    for (size_t i = 0; i < nov; i++) {
      distanceSum += visited[i];
    }
    results[index] = distanceSum;
    //printf("vertice %d has closeness of %d\n", index, distanceSum);
  }
}
