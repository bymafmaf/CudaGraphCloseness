
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void hello(char *a, int *b)
{
    a[threadIdx.x] += b[threadIdx.x];
}

void call_me_maybe(char* ad, int* bd, const int csize, const int isize, char* a, int* b, int blocksize) {
  hipMalloc( (void**)&ad, csize );
  hipMalloc( (void**)&bd, isize );
  hipMemcpy( ad, a, csize, hipMemcpyHostToDevice );
  hipMemcpy( bd, b, isize, hipMemcpyHostToDevice );

  dim3 dimBlock( blocksize, 1 );
  dim3 dimGrid( 1, 1 );
  hello<<<dimGrid, dimBlock>>>(ad, bd);
  hipMemcpy( a, ad, csize, hipMemcpyDeviceToHost );
  hipFree( ad );
}
